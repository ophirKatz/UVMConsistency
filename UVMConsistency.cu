
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#define CUDA_CHECK(f) do {                                                                \
  hipError_t e = f;                                                                      \
  if (e != hipSuccess) {                                                                 \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
      exit(1);                                                                            \
  }                                                                                       \
} while (0)

namespace UVMConsistency {

#define UVMSPACE      volatile

#define START         0
#define GPU_START     1
#define CPU_LOAD      2
#define GPU_FINISH    4
#define FINISH        5

#define NUM_SHARED 100

typedef unsigned long long int ulli;

__global__ void GPU_UVM_Writer_Kernel(UVMSPACE int *arr, UVMSPACE int *finished) {
  // Wait for CPU
  while (*finished != GPU_START);
  
  // Loop and execute writes on shared memory page - sequentially
  for (int i = 0; i < NUM_SHARED; i++) {
    // For Consistency Check
    arr[i] = 1;
     __threadfence_system();
  }
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH;

  // Wait for CPU to finish
  while (*finished != FINISH);
}

const long V = 1L << 32;

class Consistency {
private:	// Constructor & Destructor
  Consistency() {
    CUDA_CHECK(hipMallocManaged(&arr, sizeof(int) * NUM_SHARED));
    memset((void *) arr, 0, sizeof(int) * NUM_SHARED);

    CUDA_CHECK(hipMallocManaged(&finished, sizeof(int)));
    memset((void *) finished, START, sizeof(int));

    // Writing all the changes of UM to GPU
    __sync_synchronize();
  }

  ~Consistency() {
    CUDA_CHECK(hipFree((int *) arr));
    CUDA_CHECK(hipFree((int *) finished));
  }
  
private:	// Logic
  bool is_arr_full() {
    int count = 0;
    for (int i = 0; i < NUM_SHARED; i++) {
      count += arr[i];
    }
    return count == NUM_SHARED;
  }

  bool check_consistency(UVMSPACE long *arr) {
    // Read shared memory page - sequentially
    for (int i = 0; i < NUM_SHARED - 1; i += 2) {
      long value = arr[i];	// Will be [00000000;00000001] if arr[i] == 0 and arr[i + 1] == 1
			if (value == V) {
			::std::cout << "arr[i] = " << ((int *) &value)[0] << "arr[i + 1] = " << ((int *) &value)[1] << ::std::endl;

      // if (value == V) {  // arr[i] == 0 and arr[i + 1] == 1  ==> Inconsistency
        return true;
      }
    }
    return false;
  }
  
  void launch_task() {
    // Start GPU task
    GPU_UVM_Writer_Kernel<<<1,1>>>(arr, finished);

    // GPU can start
    *finished = GPU_START;
  }

  void check_consistency() {
    // While writes have not finished
    while (!is_arr_full()) {
      // Check if an inconsistency exists in the array
      if (check_consistency((long *) arr)) {
        ::std::cout << "Found Inconsistency !" << ::std::endl;
        return;
      }
    }
    ::std::cout << "No Inconsistency Found" << ::std::endl;
  }

  void finish_task() {
    while (*finished != GPU_FINISH);
    // Task is over
    *finished = FINISH;

    CUDA_CHECK(hipDeviceSynchronize());
  }
    
public:
  static void start() {
    Consistency consistency;
    // Start kernel
    consistency.launch_task();

    // Check GPU consistency
    consistency.check_consistency();

    // Finish task for CPU and GPU
    consistency.finish_task();
  }
private:
  UVMSPACE int *arr;
  UVMSPACE int *finished;
};

} // UVMConsistency namespace

int main() {
  UVMConsistency::Consistency::start();

  return 0;
}
