
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>
#include <iostream>
#include <math.h>
#include <bitset>

#define CUDA_CHECK(f) do {                                                                \
  hipError_t e = f;                                                                      \
  if (e != hipSuccess) {                                                                 \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
      exit(1);                                                                            \
  }                                                                                       \
} while (0)

namespace UVMConsistency {

#define UVMSPACE      volatile

#define START         0
#define GPU_START     1
#define CPU_LOAD      2
#define GPU_FINISH    4
#define FINISH        5

#define NUM_SHARED 100

typedef unsigned long long int ulli;

__global__ void GPU_UVM_Writer_Kernel(UVMSPACE int *arr, UVMSPACE int *finished) {
  // Wait for CPU
  while (*finished != GPU_START);
  
  // Loop and execute writes on shared memory page - sequentially
  for (int i = 0; i < NUM_SHARED; i++) {
    // For Inconsistency
    arr[i] = 1;
    __threadfence_system();
  }
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH;

  // Wait for CPU to finish
  while (*finished != FINISH);
}

class Consistency {
private:	// Constructor & Destructor
  Consistency() {
    CUDA_CHECK(hipMallocManaged(&arr, sizeof(int) * NUM_SHARED));
    memset((void *) arr, 0, sizeof(int) * NUM_SHARED);

    CUDA_CHECK(hipMallocManaged(&finished, sizeof(int)));
    memset((void *) finished, START, sizeof(int));

    // Writing all the changes of UM to GPU
    __sync_synchronize();
  }

  ~Consistency() {
    CUDA_CHECK(hipFree((int *) arr));
    CUDA_CHECK(hipFree((int *) finished));
  }
  
private:	// Logic
  bool is_arr_full() {
    int count = 0;
    for (int i = 0; i < NUM_SHARED; i++) {
      count += arr[i];
    }
    return count == NUM_SHARED;
  }

  bool check_consistency(UVMSPACE int *arr) {
    // Read shared memory page - sequentially
    for (int i = 0; i < NUM_SHARED - 1; i++) {
      int v2 = arr[i + 1];
      int v1 = arr[i];

      if (arr[i + 1] > arr[i]) {  // arr[i] == 0 and arr[i + 1] == 1  ==> Inconsistency
        return true;
      }
    }
    return false;
  }
  
  void launch_task() {
    // Start GPU task
    GPU_UVM_Writer_Kernel<<<1,1>>>(arr, finished);

    // GPU can start
    *finished = GPU_START;
  }

  void check_consistency() {
    // While writes have not finished
    while (!is_arr_full()) {
      // Check if an inconsistency exists in the array
      if (check_consistency(arr)) {
        ::std::cout << "Found Inconsistency !" << ::std::endl;
        return;
      }
    }
    ::std::cout << "No Inconsistency Found" << ::std::endl;
  }

  void finish_task() {
    while (*finished != GPU_FINISH);
    // Task is over
    *finished = FINISH;

    CUDA_CHECK(hipDeviceSynchronize());
  }
    
public:
  static void start() {
    Consistency consistency;
    // Start kernel
    consistency.launch_task();

    // Check GPU consistency
    consistency.check_consistency();

    // Finish task for CPU and GPU
    consistency.finish_task();
  }
private:
  UVMSPACE int *arr;
  UVMSPACE int *finished;
};

} // UVMConsistency namespace

int main() {
  UVMConsistency::Consistency::start();

  return 0;
}
