
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <thread>
#include <assert.h>
#include <iostream>

#include "hip/hip_runtime_api.h"

#define CUDA_CHECK(f) do {                                                                \
  hipError_t e = f;                                                                      \
  if (e != hipSuccess) {                                                                 \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
      exit(1);                                                                            \
  }                                                                                       \
} while (0)

namespace UVMConsistency {

#define UVMSPACE      volatile

#define START         0
#define GPU_START     1
#define CPU_LOAD      2
#define GPU_HOLD      3
#define CPU_HOLD      4
#define GPU_HOLD_DONE      55
#define CPU_HOLD_DONE      56
#define GPU_FINISH    5
#define FINISH        6

#define PAGE_SIZE     64 * 1024          // This is the size of a memory page in the tested GPU system [64K]
#define NUM_SHARED    ((2 * (PAGE_SIZE)) / sizeof(int))    // So the array will span at-least 2 memory pages

#define NUM_BLOCKS  1

typedef unsigned long long int ulli;

__global__ void GPU_UVM_Writer_Kernel(UVMSPACE int *kernel_arr, UVMSPACE int *kernel_finished) {
  UVMSPACE int *arr = kernel_arr + blockIdx.x * NUM_SHARED;
  UVMSPACE int *finished = kernel_finished + blockIdx.x;
  
  // Wait for CPU
  while (*finished != GPU_START);

  // Loop and execute writes on shared memory page - sequentially
  for (int i = 0; i < NUM_SHARED; i++) {
    // For Consistency Check
		if (i * sizeof(int) == PAGE_SIZE) {
			*finished = CPU_HOLD;
			while (*finished != GPU_HOLD); // { printf("[kernel]	while finished != GPU_HOLD\n"); }
		}

    arr[i] = 1;	// Write

    __threadfence_system();
		if (i * sizeof(int) == PAGE_SIZE) {
			while (*finished != CPU_HOLD_DONE);
		}
  }
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH;

  // Wait for CPU to finish
  // while (*finished != FINISH);
}

class Consistency {
private:	// Constructor & Destructor
  Consistency() {
    CUDA_CHECK(hipMallocManaged(&arr, sizeof(int) * NUM_SHARED * NUM_BLOCKS));
    memset((void *) arr, 0, sizeof(int) * NUM_SHARED * NUM_BLOCKS);

    CUDA_CHECK(hipMallocManaged(&finished, sizeof(int) * NUM_BLOCKS));
    memset((void *) finished, START, sizeof(int) * NUM_BLOCKS);

    // Writing all the changes of UM to GPU
    __sync_synchronize();
  }

  ~Consistency() {
    CUDA_CHECK(hipFree((int *) arr));
    CUDA_CHECK(hipFree((int *) finished));
  }
  
private:	// Logic
  bool is_arr_full(UVMSPACE int *arr) const {
    int count = 0;
    for (int i = 0; i < NUM_SHARED; i++) {
      count += arr[i];
    }
    return count == NUM_SHARED;
  }

  bool check_consistency_on_arr(UVMSPACE long *arr) const {
    // Read shared memory page - sequentially
		static const long maxLong = 4294967296L;
    for (int i = 0; i < NUM_SHARED - 1; i++) {
			if ((i * sizeof(int)) == PAGE_SIZE) {
				printf("[cpu]	times in if %d\n", *finished);
				while (*finished != CPU_HOLD); // { printf("[CPU]	while finished != CPU_HOLD   finished = %d\n", *finished); }
				*finished = GPU_HOLD;
			}
      long value = *((long *) (arr + i));

      if (value == maxLong) {  // arr[i] == 0 and arr[i + 1] == 1  ==> Inconsistency
        return true;
      }
			if ((i * sizeof(int)) == PAGE_SIZE) {
				*finished = CPU_HOLD_DONE;
			}
    }
    return false;
  }
  
  void launch_task() {
    // Start GPU task
    GPU_UVM_Writer_Kernel<<<NUM_BLOCKS,1>>>(arr, finished);
  }

  void check_consistency(UVMSPACE int *arr, UVMSPACE int *finished) const {
    // GPU can start
    *finished = GPU_START;

    // While writes have not finished
    while (!is_arr_full(arr)) {
      // Check if an inconsistency exists in the array
      if (check_consistency_on_arr((long *) arr)) {
        ::std::cout << "Found Inconsistency !" << ::std::endl;
        return;
      }
    }
    ::std::cout << "No Inconsistency Found" << ::std::endl;

    // Wait for GPU
    while (*finished != GPU_FINISH);
  }

  void finish_task() {
    CUDA_CHECK(hipDeviceSynchronize());
  }
    
public:
  static void handle_threads(const Consistency &consistency) {
    ::std::vector<std::thread> threads;
		for (int i = 0; i < NUM_BLOCKS; i++) {
      threads.push_back(
        ::std::thread(
          &Consistency::check_consistency,
          &consistency,
          consistency.arr + (i * NUM_SHARED),
          consistency.finished + i
        )
      );
    }

    for (auto& thread : threads) {
      thread.join();
    }
  }
  
  static void start() {
		Consistency consistency;

    // Start kernel
		hipProfilerStart();
    consistency.launch_task();
		hipProfilerStop();

    // Check GPU consistency
    handle_threads(consistency);

		// Finish task for CPU and GPU
		consistency.finish_task();
  }
private:
  UVMSPACE int *arr;
  UVMSPACE int *finished;
};

} // UVMConsistency namespace

int main() {
	UVMConsistency::Consistency::start();

  return 0;
}
