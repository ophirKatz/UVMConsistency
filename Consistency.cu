

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>
#include <iostream>
#include <math.h>
#include <bitset>

#define CUDA_CHECK(f) do {                                                                \
  hipError_t e = f;                                                                      \
  if (e != hipSuccess) {                                                                 \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
      exit(1);                                                                            \
  }                                                                                       \
} while (0)

#define ONLY_THREAD if (threadIdx.x == 0)

#define OUT
#define UVMSPACE      volatile


namespace UVMConsistency {


#define START         0
#define GPU_START     1
#define GPU_FINISH    2
#define FINISH        3

#define NUM_SHARED 10

typedef unsigned long long int ulli;


class BitManipulation {
public:
  __device__ static void set_bit(UVMSPACE ulli *mask, int index) {
    *mask = *mask | (1UL << index);
  }

  static int count_set_bits(ulli number) {
    return std::bitset<sizeof(ulli)>(number).count();
  }

  static ulli get_difference(ulli num1, ulli num2) {
    return num1 ^ num2;
  }

  static int get_first_set_bit_index(ulli number) {
    return __builtin_clz(number);
  }
};

class SharedUnit {
private:
  static void *allocate(size_t size) {
    void *ptr;
    CUDA_CHECK(hipMallocManaged(&ptr, size));
    CUDA_CHECK(hipDeviceSynchronize());
    return ptr;
  }

public:

  static void initialize_unit(UVMSPACE SharedUnit *unit, int index, int value) {
    unit->index = index;
    unit->value = value;
    __sync_synchronize();
  }

  SharedUnit() : index(0), value(0) {}

  // Memory management
  void *operator new(size_t size) {
    return allocate(size);
  }

  void *operator new[](size_t size) {
    return allocate(size);
  }

  void operator delete[](void *ptr) {
    CUDA_CHECK(hipFree(ptr));
  }

  // Properties
  UVMSPACE int index;
  UVMSPACE int value;
};

__device__ void increment_unit(UVMSPACE SharedUnit *unit, UVMSPACE ulli *mask) {
  atomicAdd((ulli *) &unit->value, (ulli) 1);
  BitManipulation::set_bit(mask, unit->index);
}

__global__ void UVM_increment(UVMSPACE SharedUnit *shared_units, UVMSPACE ulli *mask, UVMSPACE int *finished) {
  // Wait for CPU
  while (*finished != GPU_START);
  
  for (int i = 0; i < NUM_SHARED; i++) {
    UVMSPACE SharedUnit *unit = &shared_units[i];
    increment_unit(unit, mask);
  }
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH;

  // Wait for CPU to finish
  while (*finished != FINISH);
}

class Consistency {
private:

  Consistency() {
    shared_units = new SharedUnit[NUM_SHARED];
    for (int i = 0; i < NUM_SHARED; i++) {
      const int index = i;
      const int value = 0;
      SharedUnit::initialize_unit(shared_units + i, index, value);
    }

    CUDA_CHECK(hipMallocManaged(&finished, sizeof(int)));
    memset((void *) finished, START, sizeof(int));

    CUDA_CHECK(hipMallocManaged(&mask, sizeof(ulli)));
    memset((void *) mask, 0, sizeof(ulli));

    // Writing all the changes of UM to GPU
    __sync_synchronize();
  }

  ~Consistency() {
    delete[] shared_units;
    CUDA_CHECK(hipFree((int *) finished));
    CUDA_CHECK(hipFree((ulli *) finished));
  }

  
  // Logic
  // CPU Logic

  static int get_new_unit_changed(UVMSPACE ulli *mask, ulli compared_mask) {
    return BitManipulation::get_first_set_bit_index(
      BitManipulation::get_difference(*mask, compared_mask)
    );
  }

  void launch_task() {
    // Start GPU task
    UVM_increment<<<1,1>>>(shared_units, mask, finished);
  }

  void check_consistency() {
    ulli compared_mask = *mask;
    int last_unit_index = -1;

    // GPU can start
    *finished = GPU_START;

    while (*finished != GPU_FINISH) {
      while (*mask == compared_mask);
      int new_unit_index = Consistency::get_new_unit_changed(mask, compared_mask);
      assert (this->shared_units[new_unit_index].value != 0);
      if (last_unit_index + 1 != new_unit_index) {
        ::std::cout <<  "Error : Last unit was : "  << last_unit_index << 
                        " and current unit is : "   << new_unit_index << ::std::endl;
      }
      last_unit_index = new_unit_index;
    }
  }

  void finish_task() {
    // Task is over
    *finished = FINISH;
  }
  
public:
  static void start() {
    Consistency consistency;
    // Start kernel
    ::std::cout << "Launching kernel" << ::std::endl;
    consistency.launch_task();

    // Check GPU consistency
    ::std::cout << "Start CPU loop" << ::std::endl;
    consistency.check_consistency();

    // Finish task for CPU and GPU
    ::std::cout << "Finish task" << ::std::endl;
    consistency.finish_task();
  }
private:

  UVMSPACE SharedUnit *shared_units;
  UVMSPACE ulli *mask;
  volatile int *finished;
};

};  // namespace UVMConsistency


int main() {
  UVMConsistency::Consistency::start();
  return 0;
}

