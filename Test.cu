
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#define CUDA_CHECK(f) do {                                                                \
  hipError_t e = f;                                                                      \
  if (e != hipSuccess) {                                                                 \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
      exit(1);                                                                            \
  }                                                                                       \
} while (0)

namespace UVMConsistency {

#define UVMSPACE      volatile

#define START         0
#define GPU_START_WRITE     1
#define GPU_START_READ     2
#define CPU_LOAD      3
#define GPU_FINISH_WRITE    4
#define GPU_FINISH_READ    5
#define FINISH        6

#define NUM_SHARED 100

typedef unsigned long long int ulli;

__device__ bool is_arr_full(UVMSPACE int *arr) {
  int count = 0;
  for (int i = 0; i < NUM_SHARED; i++) {
    count += arr[i];
  }
  return count == NUM_SHARED;
}

__device__ bool check_consistency(UVMSPACE int *arr) {
  // Read shared memory page - sequentially
  for (int i = 0; i < NUM_SHARED - 1; i++) {
    if (arr[i + 1] > arr[i]) {  // arr[i] == 0 and arr[i + 1] == 1  ==> Inconsistency
      return true;
    }
  }
  return false;
}

__device__ void Reader_Thread(UVMSPACE int *arr, UVMSPACE int *finished) {
  // Wait for CPU
  while (*finished < GPU_START_READ);
  
  while (!is_arr_full(arr)) {
    // Check if an inconsistency exists in the array
    if (check_consistency(arr)) {
      printf("Found Inconsistency !\n");
      return;
    }
  }
  printf("No Inconsistency Found\n");
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH_READ;

  // Wait for CPU to finish
  while (*finished != FINISH);
}

__device__ void Writer_Thread(UVMSPACE int *arr, UVMSPACE int *finished) {
  // Wait for CPU
  while (*finished < GPU_START_WRITE);
  
  // Loop and execute writes on shared memory page - sequentially
  for (int i = 0; i < NUM_SHARED; i++) {
    // For Consistency Check
    arr[i] = 1;
    // __threadfence_system();
  }
  
  // GPU finished - CPU can finish
  *finished = GPU_FINISH_WRITE;

  // Wait for CPU to finish
  while (*finished != FINISH);
}

__global__ void GPU_UVM_Writer_Kernel(UVMSPACE int *arr, UVMSPACE int *finished) {
  if (threadIdx.x == 0) {
    Writer_Thread(arr, finished);
  } else {
    Reader_Thread(arr, finished);
  }
  __syncthreads();
}

class Consistency {
private:	// Constructor & Destructor
  Consistency() {
    CUDA_CHECK(hipMallocManaged(&arr, sizeof(int) * NUM_SHARED));
    memset((void *) arr, 0, sizeof(int) * NUM_SHARED);

    CUDA_CHECK(hipMallocManaged(&finished, sizeof(int)));
    memset((void *) finished, START, sizeof(int));

    // Writing all the changes of UM to GPU
    __sync_synchronize();
  }

  ~Consistency() {
    CUDA_CHECK(hipFree((int *) arr));
    CUDA_CHECK(hipFree((int *) finished));
  }
  
private:	// Logic
  bool is_arr_full() {
    int count = 0;
    for (int i = 0; i < NUM_SHARED; i++) {
      count += arr[i];
    }
    return count == NUM_SHARED;
  }

  bool check_consistency(UVMSPACE int *arr) {
    // Read shared memory page - sequentially
    for (int i = 0; i < NUM_SHARED - 1; i++) {
      // int v2 = arr[i + 1];
      // int v1 = arr[i];

      if (arr[i + 1] > arr[i]) {  // arr[i] == 0 and arr[i + 1] == 1  ==> Inconsistency
        return true;
      }
    }
    return false;
  }
  
  void launch_task() {
    // Start GPU task
    GPU_UVM_Writer_Kernel<<<1,2>>>(arr, finished);

    // GPU can start
    *finished = GPU_START_WRITE;
    *finished = GPU_START_READ;
  }

  // void check_consistency() {
  //   // While writes have not finished
  //   while (!is_arr_full()) {
  //     // Check if an inconsistency exists in the array
  //     if (check_consistency(arr)) {
  //       ::std::cout << "Found Inconsistency !" << ::std::endl;
  //       return;
  //     }
  //   }
  //   ::std::cout << "No Inconsistency Found" << ::std::endl;
  // }

  void finish_task() {
    while (*finished < GPU_FINISH_READ);
    // Task is over
    *finished = FINISH;

    CUDA_CHECK(hipDeviceSynchronize());
  }
    
public:
  static void start() {
    Consistency consistency;
    // Start kernel
    consistency.launch_task();

    // Check GPU consistency
    // consistency.check_consistency();

    // Finish task for CPU and GPU
    consistency.finish_task();
  }
private:
  UVMSPACE int *arr;
  UVMSPACE int *finished;
};

} // UVMConsistency namespace

int main() {
  UVMConsistency::Consistency::start();

  return 0;
}
