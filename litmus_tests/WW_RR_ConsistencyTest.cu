
#include <hip/hip_runtime.h>
#include <stdio.h>


#define GPU_START			1
#define GPU_FINISHED	2
#define FINISH				3

#define NUM_OF_TESTS	10000

__global__ void kernel(volatile int *x, volatile int *y, volatile int *finished) {
	while (*finished != GPU_START);

	for (int i = 1; i <= NUM_OF_TESTS; i++) {
		*x = i;
		*y = i;
	}

	*finished = GPU_FINISHED;

	// while (*finished != FINISH);
}

int main() {
	volatile int *x, *y, *finished;
	hipMallocManaged((void **) &x, sizeof(int));
	hipMallocManaged((void **) &y, sizeof(int));
	hipMallocManaged((void **) &finished, sizeof(int));

	memset((void *) x, 0, sizeof(int));
	memset((void *) y, 0, sizeof(int));
	memset((void *) finished, 0, sizeof(int));

	kernel<<<1,1>>>(x, y, finished);
	*finished = GPU_START;

	int p = 0, q = 0;
	for (int i = 1; i <= NUM_OF_TESTS; i++) {
		while ((p + q) == 0) {
			p = *y;
			q = *x;
		}

		// Perform test
		if ((p == i) && (q == i - 1)) {
			printf("Success\n");
			return;
		}
		// Reset for next test
		p = 0;
		q = 0;
	}
	printf("Failure\n");
		
	while (*finished != GPU_FINISHED);
	// *finished = FINISH;

	hipDeviceSynchronize();

	hipFree((int *) x);
	hipFree((int *) y);
	hipFree((int *) finished);

	return 0;
}


